#include <iostream>
#include <cassert>
#include <vector>
#include "polynomials/polynomials.h"
#include "polynomials/cuda_backend/polynomial_cuda_backend.cuh"
#include "ntt/ntt.cuh"
#include "api/bn254.h"
#include "vec_ops/vec_ops.cuh"


using namespace bn254;
using namespace polynomials;
using namespace vec_ops;

typedef Polynomial<scalar_t> Polynomial_t;

static std::unique_ptr<scalar_t[]> generate_pows(scalar_t tau, uint32_t size){
    auto vec = std::make_unique<scalar_t[]>(size);
    vec[0] = scalar_t::one();
    for (size_t i = 1; i < size; ++i) {
      vec[i] = vec[i-1] * tau;
  }
  return std::move(vec);
}

static std::unique_ptr<affine_t[]> generate_SRS(uint32_t size) {
  auto secret_scalar = scalar_t::rand_host();
  auto gen = projective_t::generator();
  auto pows_of_tau = generate_pows(secret_scalar,size);
  auto SRS = std::make_unique<affine_t[]>(size);
  for (size_t i = 0; i < size; ++i) {
      SRS[i] = projective_t::to_affine(pows_of_tau[i] * gen);
  }
  return std::move(SRS);
}

static Polynomial_t randomize_polynomial(uint32_t size)
{
  auto coeff = std::make_unique<scalar_t[]>(size);
  for (int i = 0; i < size; i++)
    coeff[i] = scalar_t::rand_host();
  return Polynomial_t::from_coefficients(coeff.get(), size);
}

void basic_commit(){
  int N = 1025;
  // generate group elements string of length N: (1, beta,beta^2....,beta^{N-1}). g
  auto SRS = generate_SRS(2*N);
  const int MAX_NTT_LOG_SIZE = 24;

  auto ntt_config = ntt::default_ntt_config<scalar_t>();
  const scalar_t basic_root = scalar_t::omega(MAX_NTT_LOG_SIZE);
  ntt::init_domain(basic_root, ntt_config.ctx);
  Polynomial_t::initialize(std::make_unique<CUDAPolynomialFactory<>>());
  
  //goal:
  //test commitment equality [(f1+f2)^2 + (f1-f2)^2 ]_1 = [4 (f1^2+ f_2^2)]_1
  //test commitment equality [(f1+f2)^2 - (f1-f2)^2 ]_1 = [4 f1 *f_2]_1

  auto f1 = randomize_polynomial(N);
  auto f2 = randomize_polynomial(N);
  //deg 2N constraints (f1+f2)^2 + (f1-f2)^2 = 4 (f1^2+ f_2^2)
  auto L1 = (f1+f2)*(f1+f2) + (f1-f2)*(f1-f2);
  auto R1 = scalar_t::from(2) * (f1*f1 + f2*f2);
  //deg 2N constraints (f1+f2)^2 - (f1-f2)^2 = 4 f1 *f_2
  auto L2 = (f1+f2)*(f1+f2) - (f1-f2)*(f1-f2);
  auto R2 = scalar_t::from(4) * f1 * f2;
     
  // extract coeff using coeff view
  auto [viewL1, sizeL1, device_idL1] = L1.get_coefficients_view();
  auto [viewL2, sizeL2, device_idL2] = L2.get_coefficients_view(); 
  auto [viewR1, sizeR1, device_idR1] = R1.get_coefficients_view();
  auto [viewR2, sizeR2, device_idR2] = R2.get_coefficients_view();
  
  //prepare to commit
  msm::MSMConfig config = msm::default_msm_config();
  //device vars
  affine_t* points_d;
  projective_t* L1c = nullptr, *R1c = nullptr, *L2c = nullptr, *R2c = nullptr;
  //host vars (for result)
  projective_t hL1{}, hL2{}, hR1{}, hR2{};

  //Allocate memory on device (points)
  hipMalloc(&points_d, sizeof(affine_t)* 2 * N);
    //Allocate memory on device (scalars)
  hipMalloc(&L1c, sizeof(projective_t)), hipMalloc(&R1c, sizeof(projective_t));
  hipMalloc(&L2c, sizeof(projective_t)),hipMalloc(&R2c, sizeof(projective_t));

  //copy SRS to device, scalars are already on device (we are using them via polynomial views)
  hipMemcpy(points_d, SRS.get(), sizeof(affine_t)* 2 * N, hipMemcpyHostToDevice);

  //msm bn254 api

  bn254_msm_cuda(viewL1.get(),points_d,N,config,L1c);
  bn254_msm_cuda(viewL2.get(),points_d,N,config,L2c);
  bn254_msm_cuda(viewR1.get(),points_d,N,config,R1c);
  bn254_msm_cuda(viewR2.get(),points_d,N,config,R2c);

  //out result and send to host
  hipMemcpy(&hL1, L1c, sizeof(projective_t), hipMemcpyDeviceToHost);
  hipMemcpy(&hL2,L2c, sizeof(projective_t), hipMemcpyDeviceToHost);
  hipMemcpy(&hR1,R1c, sizeof(projective_t), hipMemcpyDeviceToHost);
  hipMemcpy(&hR2, R2c, sizeof(projective_t), hipMemcpyDeviceToHost);

  //sanity checks
  auto affL1 = projective_t::to_affine(hL1);
  auto affR1 = projective_t::to_affine(hR1);

  auto affL2 = projective_t::to_affine(hL2);
  auto affR2 = projective_t::to_affine(hR2);

 //test commitment equality [(f1+f2)^2 + (f1-f2)^2]_1 = [4 (f_1^2+f_2^2]_1
  assert(affL1.x==affR1.x && affL1.y==affR1.y);
  std::cout << "Verified [(f1+f2)^2 + (f1-f2)^2]_1 = [4 (f_1^2+f_2^2]_1 " << std::endl;
 //test commitment equality [(f1+f2)^2 - (f1-f2)^2]_1 = [4 f_1 f_2]_1
  assert(affL2.x==affR2.x && affL2.y==affR2.y);
  std::cout << "Verified [(f1+f2)^2 - (f1-f2)^2]_1 = [4 f_1 f_2]_1 " << std::endl;
//print proof
  std::cout << "Proof: [" << affL1.x << ", " << affL1.y << "]" << std::endl;
//clear memory
hipFree(L1c),hipFree(L2c),hipFree(R1c),hipFree(R2c);
}